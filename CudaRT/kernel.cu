#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <time.h>
#include <fstream>

#include <hiprand/hiprand_kernel.h>

#include "defs.h"
#include "vec3.h"
#include "ray.h"
#include "math.h"
#include "sphere.h"
#include "hittable_list.h"
#include "camera.h"
#include "material.h"

#define SDL_MAIN_HANDLED
#include "SDL.h"

#define MAX_BOUNCE_DEPTH 50
#define SAMPLES_PER_PIXEL 50

__device__ Vec3f32 color(hiprandState* local_rand_state, HittableList** hittables, const Ray& r) {
    Ray cur_ray = r;
    Vec3f32 cur_attenuation = Vec3f32(1.0, 1.0, 1.0);
    for (int i = 0; i < 50; i++) {
        HitRecord rec;
        if ((*hittables)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            Ray scattered;
            Vec3f32 attenuation;
            if (rec.material->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return Vec3f32(0.0, 0.0, 0.0);
            }
        }
        else {
            Vec3f32 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            Vec3f32 c = lerp(Vec3f32(1.0, 1.0, 1.0), Vec3f32(0.5, 0.7, 1.0), t);
            return cur_attenuation * c;
        }
    }
    return Vec3f32(0.0, 0.0, 0.0); // exceeded recursion
}

__device__ double linear_to_gamma(double linear_component)
{
    if (linear_component > 0)
        return sqrt(linear_component);

    return 0;
}

__device__ Vec3f32 linear_to_gamma(Vec3f32 linear_vec)
{
   return Vec3f32(linear_to_gamma(linear_vec.x()), linear_to_gamma(linear_vec.y()), linear_to_gamma(linear_vec.z()));
}

__device__ Uint32 vec3_to_color(Vec3f32 color) {
    color = linear_to_gamma(color);

    color *= 255.99;

    const Uint8 R = color.r();
    const Uint8 G = color.g();
    const Uint8 B = color.b();

    return (0xFF << 24) | (R << 16) | (G << 8) | (B);
}

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int flipped_j = max_y - 1 - j;
    int pixel_index = flipped_j * max_x + i;
    const unsigned long long seed = 1984;
    const unsigned long long offset = 0;
    hiprand_init(seed, pixel_index, offset, &rand_state[pixel_index]);
}

__global__ void render(HittableList** hittables, hiprandState* rand_state, int ns, Uint32* fb, int max_x, int max_y, Camera** cam) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int flipped_j = max_y - 1 - j;
    int pixel_index = flipped_j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];

    Vec3f32 col(0., 0., 0.);
    for (int s = 0; s < ns; s++) {
        float u = float(i + random_float(&local_rand_state)) / float(max_x);
        float v = float(j + random_float(&local_rand_state)) / float(max_y);
        Ray r = (*cam)->get_ray(u, v, rand_state);
        col += color(&local_rand_state, hittables, r);
    }

    fb[pixel_index] = vec3_to_color(col / float(ns));
}

__global__ void create_world(Sphere** spheres, int num_hittables, HittableList** hittables, Camera** d_camera, int nx, int ny) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(spheres) = new Sphere(Vec3f32(0, 0, -1), 0.5, Material::lambertian(Vec3f32(0.1, 0.2, 0.5)));
        *(spheres + 1) = new Sphere(Vec3f32(0, -100.5, -1), 100, Material::lambertian((Vec3f32(0.8, 0.8, 0.0))));
        *(spheres + 2) = new Sphere(Vec3f32(1, 0, -1), 0.5, Material::metal(Vec3f32(0.8, 0.6, 0.2), 0.0));
        *(spheres + 3) = new Sphere(Vec3f32(-1, 0, -1), 0.5, Material::dieletric(1.5));
        *(spheres + 4) = new Sphere(Vec3f32(-1, 0, -1), -0.45, Material::dieletric(1.5));
        *hittables = new HittableList(spheres, num_hittables);
        Vec3f32 origin(3, 3, 2);
        Vec3f32 look_at(0, 0, -1);
        float dist_to_focus = (origin - look_at).length();
        float aperture = 2.0;
        *d_camera = new Camera(
            origin,
            look_at,
            Vec3f32(0, 1, 0),
            20.0,
            float(nx) / float(ny),
            aperture,
            dist_to_focus
        );
    }
}

__global__ void free_world(Sphere** spheres, HittableList** hittables, Camera** d_camera) {
    delete* (spheres);
    delete* (spheres+1);
    delete* hittables;
    delete* d_camera;
}

int main() {
    clock_t start, stop;

    int nx = 2400 / 2;
    int ny = 1200 / 2;

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(Vec3f32);

    const size_t surface_buffer_size = nx * ny * sizeof(Uint32);

    // Block stuff
    int tx = 8;
    int ty = 8;
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);

    if (SDL_Init(SDL_INIT_VIDEO) < 0) {
        printf("SDL could not be initialized!\n"
            "SDL_Error: %s\n",
            SDL_GetError());
        return 0;
    }
    SDL_Window* window = SDL_CreateWindow(
        "Basic C SDL project", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED,
        nx, ny, SDL_WINDOW_SHOWN);
    if (!window) {
        printf("Window could not be created!\n"
            "SDL_Error: %s\n",
            SDL_GetError());
        return 0;
    }
    SDL_Surface* surface = SDL_CreateRGBSurface(0, nx, ny, 32,
        0x00FF0000,
        0x0000FF00,
        0x000000FF,
        0xFF000000
    );
    if (!surface) {
        printf("Surface could not be created!\n"
            "SDL_Error: %s\n",
            SDL_GetError());
        return 0;
    }
    SDL_SetSurfaceBlendMode(surface, SDL_BLENDMODE_NONE);

    Uint32* surface_buffer;
    checkCudaErrors(hipMalloc(&surface_buffer, surface_buffer_size));

    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));

    render_init << <blocks, threads >> > (nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // TODO: Better create/free_world functions
    const int sphere_count = 5;
    Sphere** spheres;
    checkCudaErrors(hipMalloc(&spheres, sphere_count * sizeof(Sphere*)));
    HittableList** hittables;
    checkCudaErrors(hipMalloc(&hittables, sizeof(HittableList*)));
    Camera** d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(Camera*)));

    create_world << <1, 1 >> > (spheres, sphere_count, hittables, d_camera, nx, ny);

    bool quit = false;
    while (!quit) {
        SDL_Event e;
        while (SDL_PollEvent(&e)) {
            switch (e.type) {
                case SDL_KEYDOWN: {
                    switch (e.key.keysym.sym) {
                        case SDLK_ESCAPE: {
                            quit = true;
                            break;
                        }
                        default: {
                            break;
                        }
                    }
                    break;
                }
                case SDL_QUIT: {
                    quit = true;
                    break;
                }
                default: {
                    break;
                }
            }
        }

        SDL_LockSurface(surface);
        {
            start = clock();

            render << <blocks, threads >> > (
                hittables,
                d_rand_state, SAMPLES_PER_PIXEL,
                surface_buffer, nx, ny,
                d_camera
            );
            checkCudaErrors(hipGetLastError());
            checkCudaErrors(hipDeviceSynchronize());
            checkCudaErrors(hipMemcpy(surface->pixels, surface_buffer, surface_buffer_size, hipMemcpyDeviceToHost));

            stop = clock();
            double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
            std::cerr << "took " << timer_seconds << " seconds.\n";
        }
        SDL_UnlockSurface(surface);

        if (SDL_BlitScaled(surface, nullptr, SDL_GetWindowSurface(window), nullptr))
        {
            printf("SDL_BlitScaled %s", SDL_GetError());
            exit(1);
        }

        SDL_UpdateWindowSurface(window);
    }

    // Cleanup
    checkCudaErrors(hipDeviceSynchronize());
    free_world << <1, 1 >> > (spheres, hittables, d_camera);
    checkCudaErrors(hipFree(surface_buffer));
    checkCudaErrors(hipFree(spheres));

    SDL_DestroyWindow(window);
    SDL_Quit();

    return 0;
}